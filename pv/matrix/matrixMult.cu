/**
 * Simple CUDA application template.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN( value ) {							\
	hipError_t err = value;									\
	if( err != hipSuccess ) {									\
		fprintf( stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(err), __LINE__, __FILE__ );	\
		exit( 1 );												\
	} }

#define MATRIX_SIZE 10
#define BLOCK_SIZE 2

struct Matrix{
    int width;
    int height;
    float *cell;
};


__global__ void matrixMult(Matrix A, Matrix B, Matrix C){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  float val = 0;
//   printf("Row: %d  Col: %d\n", row, col);

  if (row < A.width && col < A.height){
      for(int i = 0; i < A.width; i++){
        val += A.cell[row * A.width + i] * B.cell[i * B.width + col];
//         printf(" Val: %f ", val);
        C.cell[row * C.width + col] = val;
      }
//       printf("\n");
  }
}


int main(int argc, char **argv) {
  // Alloc:
  struct Matrix A = {
    MATRIX_SIZE,
    MATRIX_SIZE,
    NULL
  };
  struct Matrix B = {
    MATRIX_SIZE,
    MATRIX_SIZE,
    NULL
  };
  struct Matrix C = {
    MATRIX_SIZE,
    MATRIX_SIZE,
    NULL
  };

  float *in_A = new float[A.width * A.height];
  for(int i = 0; i < A.width * A.height; i++){
    in_A[i] = i + 1;
  }

  float *in_B = new float[B.width * B.height];
  for(int i=0; i < B.width * B.height; i++){
    in_B[i] = i + 1;
  }

  float *out_C = new float[C.width * C.height];

  CUDA_CHECK_RETURN(hipMalloc(&A.cell, sizeof(float) * A.width * A.height));
  CUDA_CHECK_RETURN(hipMalloc(&B.cell, sizeof(float) * B.width * B.height));
  CUDA_CHECK_RETURN(hipMalloc(&C.cell, sizeof(float) * C.width * C.height));

  CUDA_CHECK_RETURN(hipMemcpy(A.cell, in_A, sizeof(float) * A.width * A.height, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(B.cell, in_B, sizeof(float) * B.width * B.height, hipMemcpyHostToDevice));

  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid((C.width + dimBlock.x - 1) / dimBlock.x, (C.height + dimBlock.y - 1) / dimBlock.y);

  matrixMult<<<dimGrid, dimBlock>>>(A, B, C);
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  CUDA_CHECK_RETURN(hipMemcpy(out_C, C.cell, sizeof(float) * C.width * C.height, hipMemcpyDeviceToHost));

  /*for(int r = 0; r < C.height; ++r){
    for(int c = 0; c < C.width; ++c){
      printf("%d  ", out_C[r * C.width + c]);
    }
    printf("\n");
  }*/

  CUDA_CHECK_RETURN(hipFree(A.cell));
  CUDA_CHECK_RETURN(hipFree(B.cell));
  CUDA_CHECK_RETURN(hipFree(C.cell));

  delete [] in_A;
  delete [] in_B;
  delete [] out_C;

  return 0;
}
