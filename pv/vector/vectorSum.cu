/**
 * Simple CUDA application template.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>
//#include <chrono>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN( value ) {							\
	hipError_t err = value;									\
	if( err != hipSuccess ) {									\
		fprintf( stderr, "Error %s at line %d in file %s\n",	\
				hipGetErrorString(err), __LINE__, __FILE__ );	\
		exit( 1 );												\
	} }

#define VECTOR_SIZE 256
#define BLOCK_SIZE 64

__global__ void vectorSum(int *A, int *B, int *C, int length){
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if(i < length)
		C[i] = A[i] + B[i];
}


int main(int argc, char **argv) {
  //auto time_start = std::chrono::steady_clock::now();

  // Alloc:
  int *d_A = NULL, *d_B = NULL, *d_C = NULL;
  int *h_A = NULL, *h_B = NULL, *h_C = NULL;

  h_A = (int*)malloc(sizeof(int) * VECTOR_SIZE);
  h_B = (int*)malloc(sizeof(int) * VECTOR_SIZE);
  h_C = (int*)malloc(sizeof(int) * VECTOR_SIZE);

  for(int i = 0; i < VECTOR_SIZE; ++i) {
    h_A[i] = i;
    h_B[i] = i;
    h_C[i] = 0; // Because output.
  }

  CUDA_CHECK_RETURN(hipMalloc(&d_A, sizeof(int) * VECTOR_SIZE));
  CUDA_CHECK_RETURN(hipMalloc(&d_B, sizeof(int) * VECTOR_SIZE));
  CUDA_CHECK_RETURN(hipMalloc(&d_C, sizeof(int) * VECTOR_SIZE));

  CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, sizeof(int) * VECTOR_SIZE, hipMemcpyHostToDevice));
  CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, sizeof(int) * VECTOR_SIZE, hipMemcpyHostToDevice));

  int GRID_SIZE = (VECTOR_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
  vectorSum<<<GRID_SIZE, BLOCK_SIZE>>>(d_A, d_B, d_C, VECTOR_SIZE);

  // Await:
  CUDA_CHECK_RETURN(hipDeviceSynchronize());
  CUDA_CHECK_RETURN(hipMemcpy(h_C, d_C, sizeof(int) * VECTOR_SIZE, hipMemcpyDeviceToHost));

  for(int i = 0; i < VECTOR_SIZE; i++)
  {
    printf("Inputs: %d, %d  Output: %d\n", h_A[i], h_B[i], h_C[i]);
  }

  // Free all:
  free(h_A);
  free(h_B);
  free(h_C);
  CUDA_CHECK_RETURN(hipFree(d_A));
  CUDA_CHECK_RETURN(hipFree(d_B));
  CUDA_CHECK_RETURN(hipFree(d_C));

  //auto time_end = std::chrono::steady_clock::now();
  //std::chrono::duration<double> time_total = time_end - time_start;
  //printf("Elapsed seconds: %d\n", time_total.count());

  return 0;
}
